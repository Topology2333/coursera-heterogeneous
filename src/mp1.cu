#include "hip/hip_runtime.h"
// MP 1
#include <cstdio>
#include <wb.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
  // 计算当前线程的索引
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // 检查索引是否越界
  if (idx < len) {
    out[idx] = in1[idx] + in2[idx]; // 执行向量加法

    // if(idx == 400)
    //    printf("%d: %f + %f\n", idx, in1[idx], in2[idx]);
  }
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput1 = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostInput2 = (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  printf("inputLength * sizeof(float) = %ld\n", inputLength * sizeof(float));
  hipMalloc((void **)&deviceInput1, inputLength * sizeof(float));
  hipMalloc((void **)&deviceInput2, inputLength * sizeof(float));
  hipMalloc((void **)&deviceOutput, inputLength * sizeof(float));

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceInput1, hostInput1, inputLength * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, inputLength * sizeof(float),
             hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  int blockSize = 256;
  int gridSize = (inputLength + blockSize - 1) / blockSize;

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  vecAdd<<<gridSize, blockSize>>>(deviceInput1, deviceInput2, deviceOutput,
                                  inputLength);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, inputLength * sizeof(float),
             hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}
